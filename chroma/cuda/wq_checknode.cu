#include "hip/hip_runtime.h"
//-*-c++-*-
#include "geometry_types.h"
#include "wq_intersect_bvh.h"

extern "C"
{

__global__ 
void checknode( const int max_loops, 
		// Photon info [index by photon id]
		float3 *positions, float3 *directions, uint* current_node, uint* test_node, int* last_result,
		// Geometry    [indexed by node id]
		const int nnodes, uint4* nodes, uint* node_parent, uint* node_first_daughter, uint* node_sibling, uint* node_aunt,
		const float3 world_origin, const float world_scale,
		// work queue  [indexed slot]
		const int queue_size, int* queue_photon_index, int* queue_slot_flag, const int first_empty_slot,
		// workgroup variables [indexed by local id]
		const int workgroup_size, const int max_nodes_can_store, 
		const int loaded_node_start_index, const int loaded_node_end_index, int* out_wavefront_start, int* out_wavefront_end
		) {
  int localid = threadIdx.x;
  //int groupid = blockIdx.x;

  __shared__ uint pop_pos;
  __shared__ uint push_pos;
  __shared__ uint nodefront_min_index;
  __shared__ uint nodefront_max_index;
  __shared__ uint requestnode_min;
  __shared__ uint requestnode_max;
  __shared__ bool transfer_nodes;
  __shared__ bool bail;
  __shared__ uint iloop;

  extern __shared__ uint shared_mem_block[];
  uint4* workgroup_nodes   = (uint4*)&shared_mem_block[0];
  uint* workgroup_daughter = &shared_mem_block[4*max_nodes_can_store];
  uint* workgroup_sibling  = &shared_mem_block[5*max_nodes_can_store];
  uint* workgroup_aunt     = &shared_mem_block[6*max_nodes_can_store];
  int* workgroup_photons        = (int*)&shared_mem_block[7*max_nodes_can_store];
  int* workgroup_current_node   = (int*)&shared_mem_block[7*max_nodes_can_store + workgroup_size];
  int* workgroup_tested_node    = (int*)&shared_mem_block[7*max_nodes_can_store + 2*workgroup_size];
  

  // initialize local variables
  if ( localid == 0) {
    pop_pos = 0;
    push_pos = (uint)first_empty_slot;
    nodefront_min_index = loaded_node_start_index;
    nodefront_max_index = loaded_node_end_index;
    requestnode_min = 0;
    requestnode_max = 1;
    iloop = 0;
    bail = false;
  }

  __syncthreads();

  // WORK QUEUE LOOP STARTS HERE
  int thread_iloop = iloop;
  int queue_index;

  while (thread_iloop<max_loops) {

    // warp fill
    // all work items load requested photon and node
    queue_index = pop_pos + localid; // this can overrun!
    if ( queue_index < queue_size && queue_slot_flag[queue_index]==1 ) {
      atomicExch( queue_slot_flag + queue_index,  0); // pop this slot
      workgroup_photons[ localid ]       = queue_photon_index[ queue_index ]; // queue needs to be filled by cpu before launching kernel
      queue_photon_index[ queue_index ] = -1;
      workgroup_current_node[ localid ]  = current_node[ workgroup_photons[ localid ] ];
      workgroup_tested_node[ localid ]   = test_node[ workgroup_photons[ localid ] ];
    }
    else if ( queue_index>=queue_size && queue_slot_flag[queue_index-queue_size]==1) {
      atomicExch( queue_slot_flag + queue_index-queue_size,  0); // pop this slot
      workgroup_photons[ localid ]       = queue_photon_index[ queue_index-queue_size ]; // queue needs to be filled by cpu before launching kernel
      queue_photon_index[ queue_index-queue_size ] = -1;
      workgroup_current_node[ localid ]  = current_node[ workgroup_photons[ localid ] ];
      workgroup_tested_node[ localid ]   = test_node[ workgroup_photons[ localid ] ];
    }
    else {
      workgroup_photons[ localid ] = -1;
      workgroup_current_node[ localid ] = -1;
      workgroup_tested_node[ localid ] = -1;
    }

    __syncthreads();

    // pop last 16
    if ( localid == 0) {
      int next_pos = pop_pos + (uint)workgroup_size;
      transfer_nodes = false;
      // move queue position
      if ( next_pos >= queue_size ) 
	next_pos = next_pos-queue_size;
      pop_pos = next_pos;
    }

    __syncthreads();

    /* // ------------------------------------------------------ */
    /* // for debug (checks pop above) */
    /* current_node[ workgroup_photons[ localid ] ] = localid; */
    /* if ( localid==0 ) { */
    /*   iloop += 1; */
    /*   // push back onto queue */
    /*   for (int i=0; i<workgroup_size; i++) { */
    /* 	queue_photon_index[ push_pos ] = workgroup_photons[ i ]; */
    /* 	atomicExch( queue_slot_flag+push_pos,  1); */
    /* 	push_pos += 1; */
    /* 	if ( push_pos>=(uint)queue_size ) */
    /* 	  push_pos = 0; */
    /*   } */
    /* } */
    /* barrier( CLK_LOCAL_MEM_FENCE ); */
    /* thread_iloop = iloop;     */
    /* continue; */
    /* // ------------------------------------------------------ */

    // thread zero, polls range of nodes to get
    if ( localid == 0) {

      requestnode_min = workgroup_tested_node[ 0 ];
      for (int i=0; i<workgroup_size; i++) {
	if ( workgroup_photons[ i ]>=0 ) { // we ignore non-filled threads
	  //requestnode_min = min( requestnode_min, (uint)workgroup_current_node[ i ] );
	  requestnode_min = min( requestnode_min, (uint)workgroup_tested_node[ i ] );
	  requestnode_max = max( requestnode_max, (uint)workgroup_tested_node[ i ] );
	}
      }

      // easy scenario, we can fit all requested nodes (or more) into shared memory

      if ( (requestnode_max-requestnode_min)<=(uint)max_nodes_can_store ) {
	requestnode_max = requestnode_min+max_nodes_can_store;
	// if we don't have the nodes we need, schedule a transfer, update the front
	if ( requestnode_min<nodefront_min_index || requestnode_max>nodefront_max_index ) {
	  transfer_nodes = true;
	  nodefront_min_index = requestnode_min;
	  nodefront_max_index = requestnode_max;
	}
      }
      else {
	// hard scenario, we can't load them all in. 

	// fancy option
	// thread zero could keep pushing and popping photons until it collects enough work items
	//requestnode_max = requestnode_min+max_nodes_can_store;
	/* for (int i=0; i<workgroup_size; i++) { */
	/*   if ( workgroup_tested_node[ i ]>requestnode_max ) { */
	/*     // push photon item back onto queue */
	/*     queue_photon_index[ push_pos ] = workgroup_photons[ i ]; */
	/*     atomicExch( queue_slot_flag+push_pos,  1); */
	/*     push_pos += 1; */
        /*     if ( push_pos>=(uint)queue_size ) */
        /*       push_pos = 0; */

	/*     // pop end of photon */
	/*     workgroup_photons[ i ] = queue_photon_index[ pop_pos ]; */
	/*     workgroup_current_node[ i ] = current_node[ workgroup_photons[ i ] ]; */
	/*     workgroup_tested_node[ localid ] = test_node[ workgroup_photons[ i ] ]; */
	/*     atomicExch( queue_slot_flag+pop_pos,  0); */
	/*     queue_photon_index[ pop_pos ] = -2; */
	/*     pop_pos += 1; */
	/*     if ( pop_pos>=(uint)queue_size ) */
        /*       pop_pos = 0; */
	/*   } */
	/* } */

	// cave man option
	// pull in what we need, but let other threads go to global memory...	
	transfer_nodes = true;
	nodefront_min_index = requestnode_min;
	nodefront_max_index = requestnode_min+max_nodes_can_store;
      }	  
    } //end of thread-0

    // workgroup works together to load nodes into local memory
    int num_blocks = (nodefront_max_index-nodefront_min_index)/workgroup_size;
    if ( (nodefront_max_index-nodefront_min_index)%workgroup_size!=0 )
      num_blocks++;

    __syncthreads();
    if ( bail )
      break;

    if ( transfer_nodes ) {
      for (int iblock=0; iblock<num_blocks; iblock++ ) {
    	int local_inode = iblock*workgroup_size + localid;
    	int global_inode = nodefront_min_index + iblock*workgroup_size + localid;
    	if ( local_inode < max_nodes_can_store && global_inode<nnodes ) {
    	  workgroup_nodes[ local_inode ] = nodes[ global_inode ];
	  workgroup_daughter[ local_inode ] = node_first_daughter[ global_inode ];
	  workgroup_sibling[ local_inode ]  = node_sibling[ global_inode ];
	  workgroup_aunt[ local_inode ]     = node_aunt[ global_inode ];
	}
      }
    }
    __syncthreads();

    // -- Now we finally get to testing intersections --
    
    if ( workgroup_photons[ localid ]>=0 ) { // we ignore non-filled threads

      // get photon information
      float3 photon_pos = positions[ workgroup_photons[ localid ] ];  // global access
      float3 photon_dir = directions[ workgroup_photons[ localid ] ]; // global access
      uint local_test_nodeid    = workgroup_tested_node[ localid ]  - nodefront_min_index; // zero index
      //uint local_current_nodeid = workgroup_current_node[ localid ] - nodefront_min_index; // zero index

      // get and unpack node
      Node node_struct;
      uint4 workitem_node;
      if ( local_test_nodeid<max_nodes_can_store )
	workitem_node = workgroup_nodes[ local_test_nodeid ]; // get the tested node
      else
	workitem_node = nodes[ workgroup_tested_node[ localid ] ]; // non-localized, warped global access :(
      uint3 lower_int = make_uint3(workitem_node.x & 0xFFFF, workitem_node.y & 0xFFFF, workitem_node.z & 0xFFFF);
      uint3 upper_int = make_uint3(workitem_node.x >> 16, workitem_node.y >> 16, workitem_node.z >> 16);
      float3 flower = make_float3( lower_int.x, lower_int.y, lower_int.z );
      float3 fupper = make_float3( upper_int.x, upper_int.y, upper_int.z );
      node_struct.lower = world_origin + flower * world_scale;
      node_struct.upper = world_origin + fupper * world_scale;
      node_struct.child = workitem_node.w & ~NCHILD_MASK;
      node_struct.nchild = workitem_node.w >> CHILD_BITS;
      
      int intersects = intersect_internal_node( photon_pos, photon_dir, node_struct );
      last_result[ workgroup_photons[ localid ] ] = intersects;
      
      if ( intersects ) {
	// passes. update current node to test node.  set test node as first daughter of new node
	uint next_daughter;
	if ( local_test_nodeid<max_nodes_can_store )
	  next_daughter = workgroup_daughter[ local_test_nodeid ];
	else
	  next_daughter = node_first_daughter[ workgroup_tested_node[ localid ] ];
	// store next nodes in local space first (later we will push the info into global memory
	workgroup_current_node[ localid ] =  workgroup_tested_node[ localid ];
	workgroup_tested_node[ localid ] = next_daughter;
      }
      else {
	// does not pass.  check sibling of tested node.
	uint sibling;
	uint aunt;
	if ( local_test_nodeid<max_nodes_can_store ) {
	  sibling = workgroup_sibling[ local_test_nodeid ];
	  aunt    = workgroup_aunt[ local_test_nodeid ];
	}
	else {
	  sibling = node_sibling[ workgroup_tested_node[ localid ] ];
	  aunt    = node_aunt[ workgroup_tested_node[ localid ] ];
	}
	// current node is unchanged
	workgroup_tested_node[ localid ] = sibling;
	if ( workgroup_tested_node[ localid ]==0 )
	  workgroup_tested_node[ localid ] = aunt;
      }
      
      __syncthreads();
      
      // Now thread 0 pushes threads to end of queue for next step, if not a leaf node
      if ( localid==0 ) {
	// check each thread. if !leaf or !non-aunt, push to end of work queue
	for (int i=0; i<workgroup_size; i++) {
	  uint nchild = 0;
	  if ( workgroup_current_node[ i ]>=nodefront_min_index && workgroup_current_node[ i ]<nodefront_max_index )
	    nchild = workgroup_nodes[ workgroup_current_node[ i ]-nodefront_min_index ].w >> CHILD_BITS;
	  else
	    nchild = nodes[ workgroup_current_node[ i ] ].w >> CHILD_BITS; // outside node-front, so have to go to global memory
	  if ( nchild>0 ) {
	    // internal node
	    queue_photon_index[ push_pos ] = workgroup_photons[ i ];
	    atomicExch( queue_slot_flag+push_pos,  1);
	    push_pos += 1;
	    if ( push_pos>=(uint)queue_size )
	      push_pos = 0;
	  }
	  else {
	    
	    last_result[ workgroup_photons[ i ] ] = 2; //leaf node
	  }
	  // push to global: atomic to prevent competition with other compute units
	  atomicExch( current_node + workgroup_photons[ i ], workgroup_current_node[ i ] );
	  atomicExch( test_node + workgroup_photons[ i ],    workgroup_tested_node[ i ] );

	}
      }
      
      // For debug
/*       // assume it intersects, update photon queue */
/*       //current_node[ workgroup_photons[ localid ]  ] = workgroup_tested_node[ localid ]; */
/*       //current_node[ workgroup_photons[ localid ]  ] = workitem_node.w; */
/*       //current_node[ workgroup_photons[ localid ]  ] = local_test_nodeid; */
//      current_node[ workgroup_photons[ localid ]  ] = (uint)intersects;
    } // if valid photon

    __syncthreads();    

    if ( localid==0 ) {
      iloop += 1;
    }
    __syncthreads();
    thread_iloop = iloop;
    //barrier( CLK_LOCAL_MEM_FENCE );
  } // end of while loop

  if ( localid==0 ) {
    *out_wavefront_start = nodefront_min_index;
    *out_wavefront_end   = nodefront_max_index;
  }

  return;

}

}// end of extern C
